
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 600

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
	int i = blockIdx.x;
    DC[i] = DA[i] + DB[i];
}

int main()
{ int HA[N], HB[N], HC[N];
  int *DA, *DB, *DC;
  int i; int size = N*sizeof(int);
  hipError_t error;
  
  // reservamos espacio en la memoria global del device
  error = hipMalloc((void**)&DA, size);
  if (error != hipSuccess) printf("%s\n", hipGetErrorString(error));
  error = hipMalloc((void**)&DB, size);
  if (error != hipSuccess) printf("%s\n", hipGetErrorString(error));
  error = hipMalloc((void**)&DC, size);
  if (error != hipSuccess) printf("%s\n", hipGetErrorString(error));
  
  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  error = hipMemcpy(DA, HA, size, hipMemcpyHostToDevice);
  if (error != hipSuccess) printf("%s\n", hipGetErrorString(error));
  error = hipMemcpy(DB, HB, size, hipMemcpyHostToDevice);
  if (error != hipSuccess) printf("%s\n", hipGetErrorString(error));
  
  // llamamos al kernel (1 bloque de N hilos)
  VecAdd <<<N, 1>>>(DA, DB, DC);	// N hilos ejecutan el kernel en paralelo
  
  hipError_t error_synchro = hipGetLastError();
  hipError_t error_asynchro = hipDeviceSynchronize();
  if (error_synchro != hipSuccess) printf("Sync kernel error: %s\n", hipGetErrorString(error_synchro));
  if (error_asynchro != hipSuccess) printf("Async kernel error: %s\n", hipGetErrorString(error_asynchro));
  
  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  error = hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost);
  if (error != hipSuccess) printf("%s\n", hipGetErrorString(error));
  
  // liberamos la memoria reservada en el device
  hipFree(DA); hipFree(DB); hipFree(DC);  
  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  // esta comprobación debe quitarse una vez que el programa es correcto (p. ej., para medir el tiempo de ejecución)
  for (i = 0; i < N; i++){
    //printf("%d + %d = %d\n",HA[i],HB[i],HC[i]);
    if (HC[i]!= (HA[i]+HB[i])) 
		{
        printf("error en componente %d\n", i);
        break;}
  }
    
  return 0;
} 
